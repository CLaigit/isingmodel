/*
 *  Ising model: Halmitonian H = /sum_ij J(sigma_i)(sigma_j)
 */

/*
 *   1. Calculate the energy in the program
 *   2. Calculate the heat capacity in the program
 *   3. Add more inputs to adjust the length of lattice
 *   4. A matlab code to plot data.
 *       data format example:
 *                    position.x  position.y   spin(-1, 1)
 *       Iteattion 1:    1           4               -1
 *                       *           *                *
 *                       *           *                *
 *       Iteattion 2:    4           3                1
 *                       *           *                *
 *                       *           *                *
 *       Iteattion N:    35          76               1
 *                       *           *                *
 *                       *           *                *
 *   5. Compare the numerical value with the analytic value
 *   6. Move to 3D
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>       /* time */
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

/*
 * LATTICE_LENGTH  - length of the lattice
 * LATTICE_LENGTH  - number of element is one lattice
 * BOLTZMANN_CONST - bolzmann constant. Set to 1.
 */

#define LATTICE_LENGTH 1024
#define LATTICE_2 (LATTICE_LENGTH * LATTICE_LENGTH)
#define BOLTZMANN_CONST 1
#define N LATTICE_LENGTH

#define WARM_STEP 1e3
#define MEAS_STEP 1e3
#define WARP 1e1
#define NUM_THREAD_X 32
#define NUM_THREAD_Y 32
#define TEMPERATURE 4.0

__device__ int energy(int up, int down, int left, int right, int center);
__global__ void update(int *lattice, double beta, double *E_d, double *M_d, double *E2_d, double *M2_d, int tag, hiprandState * global_state);
__global__ void printstate(int *lattice);
__global__ void init_rand(hiprandState * global_state, unsigned long seed);


/* Setup random seed to each kernel */
__global__ void init_rand(hiprandState * global_state, unsigned long seed){
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	hiprand_init(seed, idx + idy * N, 0, &global_state[idx + idy * N]);
	__syncthreads();
}

/*
 *   update is the function to update a point
 *   1. flip a point (1 -> -1 or -1 -> 1)
 *   2. compare the energy before flip a point and after flip a point
 *   3. if the energy with flipped point is small, accept
 *   4. if the energy is larger, generate a random number pro_rand (0,1),
 *      if pro_rand < e^(-beta * delatE), aceept. else reject.
 */
__global__ void update(int* lattice, double beta, double *E_d, double *M_d, double *E2_d, double *M2_d, int tag, hiprandState * global_state){
	// Calculate the global index
	// Calculate the global index for the up, down, left, right index.
		
	// declare parameters
	int itx, ity, idx, idy, index;
	int flip, up, down, left, right, center;
	double pro_rand, deltaE, E;

	// local index
	itx = threadIdx.x;
	ity = threadIdx.y;

	// global index
	idx = blockIdx.x * blockDim.x + itx;
	idy = blockIdx.y * blockDim.y + ity;
	index = idx * N + idy;
		
	// load data into shared memory
	__shared__ int lat[32 + 2][32 + 2];
	__syncthreads();

	lat[itx+1][ity+1] = lattice[index];

	if(idx == 0){
		lat[itx][ity + 1] = lattice[index + (N - 1) * N];
	}else if(itx == 0){
		lat[itx][ity + 1] = lattice[index - N];
	}

	if(idx == N - 1){
		lat[itx + 2][ity + 1] = lattice[index - (N - 1) * N];
	}else if(itx == NUM_THREAD_X - 1){
		lat[itx + 2][ity + 1] = lattice[index + N -1];
	}

	if(idy == 0){
		lat[itx + 1][ity] = lattice[index + N - 1];
	}else if(ity == 0){
		lat[itx + 1][ity] = lattice[index - 1];
	}

	if(idy == N - 1){
		lat[itx + 1][ity + 2] = lattice[index - (N - 1)];
	}else if(ity == NUM_THREAD_X - 1){
		lat[itx + 1][ity + 2] = lattice[index + 1];
	}
		
	hiprandState local_state = global_state[idx * N + idy];
	pro_rand = hiprand_uniform(&local_state);
	global_state[idx * N + idy] = local_state;
	__syncthreads();

	// for even sites
	if((idx + idy) % 2 == 0){
   	up     = lat[itx][ity + 1];
   	down   = lat[itx + 2][ity + 1];
   	left   = lat[itx + 1][ity];
   	right  = lat[itx + 1][ity + 2];
   	center = lat[itx + 1][ity + 1];

		// Flip the center element
		flip = -center;

		// Calculate the difference between these two state
		E      = energy(up, down, left, right, center);
		deltaE = -2.0 * E;

		// If deltaE < 0 or pro_rand <= e^(-beta * deltaE), accept new value
		if (deltaE < 0 || pro_rand <= exp(- 1.0 * beta * (deltaE * 1.0))){
      lat[itx + 1][ity + 1] *= -1;
     }
	}

	// wait for even site completion
	__syncthreads();

	// for odd sites
	if((idx + idy) % 2 == 1){
		up     = lat[itx][ity + 1];
    down   = lat[itx + 2][ity + 1];
    left   = lat[itx + 1][ity];
    right  = lat[itx + 1][ity + 2];
    center = lat[itx + 1][ity + 1];
	
		// Flip the center element
		flip = -center;

		// Calculate the difference between these two state
    E      = energy(up, down, left, right, center);
		deltaE = -2.0 * E;

		// If deltaE < 0 or pro_rand <= e^(-beta * deltaE), accept new value
		if (deltaE < 0 || pro_rand <= exp(- 1.0 * beta * (deltaE * 1.0))){
			lat[itx + 1][ity + 1] *= -1;
		}
	}

	// wait for odd site completion
	__syncthreads();
			

	// store data back
	lattice[index] = lat[itx + 1][ity + 1];

	if(tag == 1){
		E_d[index] += E;
		M_d[index] += lat[itx+1][ity+1];
		E2_d[index] += E * E;
		M2_d[index] += lat[itx+1][ity+1] * lat[itx+1][ity+1];
	}
	__syncthreads();

}

/*
 *   printstate is the function to print the whole matrix.
 *   Since it prints in parallel, we also print the global
 *   index of the matrx.
 *   it prints (x, y, (1 or -1)).
 */
__global__ void printstate(int* lattice) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < N && idy < N){
		printf("%d, %d, %d\n", idx, idy, lattice[idx + idy * N]);
	}
	__syncthreads();
}

/*
 *   energy is the function used to calculate the energy between
 *   (center, up), (center, down), (center, left), (center, right)
 */
__device__ int energy(int up, int down, int left, int right, int center){
	double H;
	H = - up * center - down * center - left * center - right * center;
	return H;
}

/*
 *   Commandline inputs option
 *   1. Tempurature (T)
 *
 */
int main (int argc, char *argv[]){

	int *lattice;
	int *d_lattice;

	double *E;
	double *E_d;

	double *E2;
	double *E2_d;

	double *M;
	double *M_d;

	double *M2;
	double *M2_d;

	double T = TEMPERATURE;
 	int warmsteps = WARM_STEP;
	int nout = MEAS_STEP;
	int warp = WARP;

	int numthreadx = NUM_THREAD_X;
	int numthready = NUM_THREAD_Y;
	int numblocksX = LATTICE_LENGTH / numthreadx;
	int numblocksY = LATTICE_LENGTH / numthready;

	// First input: Tempurature. Usually between (1, 6),
	// Critical Tempurature is around 2.2
	T = argc > 1 ? atof(argv[1]) : T;
	warmsteps = argc > 2 ? atof(argv[2]) : warmsteps;
	nout = argc > 3 ? atof(argv[3]) : nout;
	warp = argc > 4 ? atof(argv[4]) : warp;

	// Define the size of lattice and energy
	const size_t bytes_lattice = LATTICE_2 * sizeof(int);
	const size_t bytes_E = LATTICE_2 * sizeof(double);
	const size_t bytes_M = LATTICE_2 * sizeof(double);

	// Allocate memory for lattice. It is a lattice^2 long array.
	// The value can only be 1 or -1.
	lattice = (int*)malloc(LATTICE_2 * sizeof(int));

	E = (double*)malloc(LATTICE_2 * sizeof(double));
	M = (double*)malloc(LATTICE_2 * sizeof(double));

	E2 = (double*)malloc(LATTICE_2 * sizeof(double));
	M2 = (double*)malloc(LATTICE_2 * sizeof(double));
		

	// initialize lattice by rand(-1, 1)
	for(int i = 0; i < LATTICE_2; i++){
		lattice[i] = 2 * (rand() % 2) - 1;
		E[i] = 0.0;
		M[i] = 0.0;
		E2[i] = 0.0;
		M2[i] = 0.0;
   }

	// Set dimensions of block and grid
	dim3 grid(numblocksX, numblocksY, 1);
	dim3 thread(numthreadx, numthready,1);

	// set up random for each kernel 
	hiprandState *global_state;
	hipMalloc(&global_state, LATTICE_2 * sizeof(hiprandState));
	init_rand<<< grid, thread >>> (global_state, unsigned(time(NULL)));

	// beta is a parameter in the probability
	double beta = 1.0 / (BOLTZMANN_CONST * 1.0) / T;

	// Allocate memoery in device and copy from host to device
	hipMalloc((void **)&d_lattice, bytes_lattice);
	hipMalloc((void **)&E_d, bytes_E);
	hipMalloc((void **)&M_d, bytes_M);

	hipMalloc((void **)&E2_d, bytes_E);
	hipMalloc((void **)&M2_d, bytes_M);

	hipMemcpy(d_lattice, lattice, bytes_lattice, hipMemcpyHostToDevice);
	hipMemcpy(E_d, E, bytes_E, hipMemcpyHostToDevice);
	hipMemcpy(M_d, M, bytes_M, hipMemcpyHostToDevice);

	hipMemcpy(E2_d, E2, bytes_E, hipMemcpyHostToDevice);
	hipMemcpy(M2_d, M2, bytes_M, hipMemcpyHostToDevice);

	// To change the buffer size of printf; otherwise it cannot print all data
	hipDeviceSetLimit(hipLimitPrintfFifoSize, N * N * sizeof(int));

//	printf("Testing for T = %2f, beta = %2f...\n", T, beta);
	
	// Warmup process
//	printf("Starting Warming Steps... \n");
	int cnt = 0;

	for (int iter = 0; iter < warmsteps; iter++){
//		printf("\r [ %f% ] ", (100.0 * cnt++) / warmsteps);
			
		update<<<grid, thread>>>(d_lattice, beta, E_d, M_d, E2_d, M2_d, 0, global_state);
		hipDeviceSynchronize();
	}
//	printf("\n");

	// Measure process
//	printf("Starting Measurement Steps... \n");
	cnt = 0;
	int cnt2 = 0;

	for (int nstep = 0; nstep < nout; nstep++){
//		printf("\r [ %f% ] ", (100.0 * cnt++) / nout);

		if(nstep % warp == 0){
			cnt2++;
     	update<<<grid, thread>>>(d_lattice, beta, E_d, M_d, E2_d, M2_d, 1, global_state);
		}else{
     	update<<<grid, thread>>>(d_lattice, beta, E_d, M_d, E2_d, M2_d, 0, global_state);
		}
		hipDeviceSynchronize();

	}
//	printf("\n");
		
	double energy = 0.0;
	double magnetization = 0.0;

	double energy2 = 0.0;
	double magnetization2 = 0.0;

	hipMemcpy(lattice, d_lattice, bytes_E, hipMemcpyDeviceToHost);

	hipMemcpy(E, E_d, bytes_E, hipMemcpyDeviceToHost);
	hipMemcpy(M, M_d, bytes_M, hipMemcpyDeviceToHost);

	hipMemcpy(E2, E2_d, bytes_E, hipMemcpyDeviceToHost);
	hipMemcpy(M2, M2_d, bytes_M, hipMemcpyDeviceToHost);
		
	for(int i = 0; i < LATTICE_2; i++){
		energy += E[i];
		magnetization += M[i];

		energy2 += E2[i];
		magnetization2 += M2[i];
	}
	
	double avg_E = energy / cnt2 / (LATTICE_2 * 1.0) / 2.0;
	double avg_M = magnetization / cnt2 / (LATTICE_2 * 1.0);
	avg_M = avg_M < 0 ? -avg_M : avg_M;

	double avg_E2 = energy2 / cnt2 / (LATTICE_2 * 1.0) / 4.0;
	double avg_M2 = magnetization2 / cnt2 / (LATTICE_2 * 1.0);

	double heat_cap = 1.0 * (avg_E2 - avg_E * avg_E) / T / T;
	double mag_sus  = 1.0 * (avg_M2 - avg_M * avg_M) / T; 

//	printf("Average energy: %5f \n", avg_E);
//	printf("Average magnetization: %5f \n", avg_M);
	printf("%5f %5f %5f %5f %5f\n", T, avg_E, avg_M, heat_cap, mag_sus);

	free(lattice);
	free(E);
	free(M);
	free(E2);
	free(M2);
	hipFree(d_lattice);
	hipFree(E_d);
	hipFree(M_d);
	hipFree(E2_d);
	hipFree(M2_d);
		
	return 0;
}
